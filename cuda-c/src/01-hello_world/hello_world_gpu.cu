
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{   
    if (threadIdx.x == 5)
        printf(".............Hello World from GPU thread %d!.............\n", threadIdx.x);
}

int main(void){
    // hello from cpu
    printf("<------------Hello World from CPU!-------------->\n");
    
    helloFromGPU <<<1, 10>>>();
   
    hipDeviceSynchronize();
    return 0;
}