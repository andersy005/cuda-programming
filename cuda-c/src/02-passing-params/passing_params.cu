#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../utils/common.h"

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(){
    int c;
    int *device_c;

    HANDLE_ERROR(hipMalloc((void**)&device_c, sizeof(int)));

    add<<<1,1>>>(2, 7, device_c);

    HANDLE_ERROR(hipMemcpy(&c,
                           device_c,
                           sizeof(int),
                           hipMemcpyDeviceToHost));

    printf(" 2 + 7 = %d\n", c);
    hipFree(device_c);

    return 0;
}