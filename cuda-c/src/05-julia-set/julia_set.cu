int main(void){
    CPUBitmap bitmap (DIM, DIM);
    unsigned char *device_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&device_bitmap, bitmap.image_size()));

    dim3 grid(DIM, DIM);

    kernel<<<grid, 1>>>(device_bitmap);

    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), device_bitmap,
                            bitmap.image_size(), hipMemcpyDeviceToHost));

    bitmap.display_and_exit();
    hipFree(device_bitmap);

    return 0; 
}